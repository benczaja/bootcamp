#include <pmt.h> // needed for PMT
#include <iostream>
#include <hip/hip_runtime.h>


int main( int argc, char *argv[] )  {

    int array_length = 10000;

    // Initialize memory on host
    size_t sizeA = sizeof(double) * array_length;

    // Allocate Host Memory
    double  *h_A = (double*) malloc(sizeA);

    // Initialize the array with values
    for (int i=0; i<array_length ; ++i) h_A[i] = double(i%3+1);

    // Initialize the Sensor
    std::unique_ptr<pmt::PMT> sensor = pmt::nvml::NVML::Create();
   
   // Read from the PMT Sensor
   pmt::State start = sensor->Read();

   // Allocate Device Memory
   double  *d_A;
   hipMalloc((void **)&d_A, sizeA);

   hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
   hipMemcpy(d_A, h_A, sizeA, hipMemcpyDeviceToHost);

   hipFree(d_A);

   // Read from the PMT Sensor
   pmt::State end = sensor->Read();  
 
  std::cout << "RESULTS-------" << std::endl;
  std::cout << "PMT (NVML) Seconds: " << pmt::PMT::seconds(start, end) << " s"<< std::endl;
  std::cout << "PMT (NVML) Joules: " << pmt::PMT::joules(start, end) << " J" << std::endl;
  std::cout << "PMT (NVML) Watts: " << pmt::PMT::watts(start, end) << " W" << std::endl;

}
